
#include <hip/hip_runtime.h>
#include <stdint.h>

#define UINT64_BITS             64
#define BLOCKS   (28*16)
#define THREADS  (64*16)  /* Must be 2^n */
#define POPCNT(a) __popcll(a)

extern "C"
{
__global__ void clear_buffers_dev(uint64_t* __restrict__ A, uint64_t* __restrict__ B, const int length)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid<length) {
    A[tid] = B[tid] = 0;
    tid += blockDim.x * gridDim.x;
  }
}
__global__ void init_dev(uint64_t* __restrict__ A, uint64_t* __restrict__ B,
			 const int nodes, const unsigned int elements)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < nodes) {
    unsigned int offset = tid*elements+tid/UINT64_BITS;
//    A[offset] = B[offset] = (0x1ULL << (tid%UINT64_BITS));
    A[offset] = B[offset] = (0x1ULL << (UINT64_BITS-tid%UINT64_BITS-1));
    tid += blockDim.x * gridDim.x;
  }
}
//__global__ static void matrix_op_dev(const uint64_t* __restrict__ A, uint64_t* __restrict__ B, const int* __restrict__ adjacency,
__global__ void matrix_op_dev(const uint64_t* __restrict__ A, uint64_t* __restrict__ B, const int* __restrict__ adjacency,
				     const int* __restrict__ num_degrees, const int nodes, const int degree, const unsigned int elements)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < nodes*elements) {
    int i = tid / elements;
    int k = tid % elements;
    uint64_t tmp = B[tid];
    for(int j=0;j<num_degrees[i];j++){
      int n = *(adjacency + i * degree + j);  // int n = adjacency[i][j];
      tmp |= A[n*elements+k];
    }
    B[tid] = tmp;
    tid += blockDim.x * gridDim.x;
  }
}
//__global__ static void popcnt_dev(const uint64_t* __restrict__ B, const int nodes, 
__global__ void popcnt_dev(const uint64_t* __restrict__ B, const int nodes, 
				  const unsigned int elements, uint64_t* __restrict__ result)
{
  __shared__ uint64_t cache[THREADS];
  int cacheIndex = threadIdx.x;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  uint64_t num = 0;
  while (tid < elements*nodes) {
    num += POPCNT(B[tid]);
    tid += blockDim.x * gridDim.x;
  }
  cache[cacheIndex] = num;
  __syncthreads();

  int i = blockDim.x/2;
  while (i != 0){
    if (cacheIndex < i)
      cache[cacheIndex] += cache[cacheIndex+i];
    __syncthreads();
    i /= 2;
  }

  if(cacheIndex == 0)
    result[blockIdx.x] = cache[0];
}
}
